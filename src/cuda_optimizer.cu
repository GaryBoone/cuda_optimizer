#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <string>

#include "adaptive_sampler.h"
#include "example.h"
#include "examples/add.h"
#include "examples/euclidian_distance.h"
#include "kernels.h"
#include "metrics.h"
#include "reporter.h"
#include "timer.h"

// Build:
// $ cmake -B build -S .
// $ cmake --build build
// Run:
// $ ./build/src/cuda_optimizer
// Test:
// $ ./build/tests/test_app

const double kRequiredPrecision = 0.35;

int CheckResult(float *y, int n) {
  int num_errors = 0;
  double max_error = 0.0;

  for (int i = 0; i < n; i++) {
    if (fabs(y[i] - 3.0f) > 1e-6) {
      num_errors++;
    }
    max_error = fmax(max_error, fabs(y[i] - 3.0f));
  }

  if (num_errors > 0) {
    std::cout << "  number of errors: " << num_errors;
  }
  if (max_error > 0.0) {
    std::cout << ",  max error: " << max_error;
  }
  return num_errors;
}

float TimeKernel(IKernel &ex, int num_blocks, int block_size) {
  CudaTimer timer;
  timer.Start();
  ex.RunKernel(num_blocks, block_size);
  timer.Stop();

  // Wait for GPU to finish before accessing on host.
  hipDeviceSynchronize();

  return timer.ElapsedMilliseconds();
}

hipDeviceProp_t HardwareInfo() {
  int num_devices = 0;
  hipGetDeviceCount(&num_devices);  // Get the number of devices
  if (num_devices == 0) {
    std::cout << "No CUDA devices found." << std::endl;
    exit(1);  // TODO(Gary): Fix.
    // return 0;
  }

  std::cout << "Number of CUDA devices: " << num_devices << std::endl;
  hipDeviceProp_t props;
  for (int i = 0; i < num_devices; i++) {
    hipGetDeviceProperties(&props, i);
    std::cout << "Device Number: " << i << std::endl;
    std::cout << "  Device name: " << props.name << std::endl;
    std::cout << "  Number of SMs: " << props.multiProcessorCount << std::endl;
    std::cout << "  Total global memory: "
              << props.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "  Compute capability: " << props.major << "." << props.minor
              << std::endl;
    std::cout << "  Maximum threads per SM: "
              << props.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "  Maximum warps: " << props.warpSize << std::endl;
    std::cout << "  Maximum threads per block: " << props.maxThreadsPerBlock
              << std::endl;
    std::cout << "  Maximum thread dimensions: (" << props.maxThreadsDim[0]
              << ", " << props.maxThreadsDim[1] << ", "
              << props.maxThreadsDim[2] << ")" << std::endl;
    std::cout << "  Amount of shared memory per SM: "
              << props.sharedMemPerMultiprocessor << " bytes" << std::endl;
    std::cout << "  Number of registers per SM: " << props.regsPerMultiprocessor
              << std::endl;
  }
  return props;
}

double Occupancy(hipDeviceProp_t props, int num_blocks, int block_size,
                 kernelFuncPtr kernel) {
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, kernel, block_size,
                                                0);
  int activeWarps = num_blocks * block_size / props.warpSize;
  assert(0 != props.warpSize);
  int maxWarps = props.maxThreadsPerMultiProcessor / props.warpSize;
  return (static_cast<double>(activeWarps) / maxWarps);
}

tl::expected<AdaptiveSampler, ErrorInfo> RepeatUntil(double required_precision,
                                                     IKernel &ex,
                                                     int num_blocks,
                                                     int block_size) {
  AdaptiveSampler stats(required_precision);
  bool skip_first = true;
  while (stats.ShouldContinue()) {
    ex.Reset();

    float time = TimeKernel(ex, num_blocks, block_size);

    if (0 != ex.CheckResults()) {
      return tl::make_unexpected(ErrorInfo(ErrorInfo::kUnexpectedKernelResult,
                                           "errors in kernel results"));
    }

    // Don't include the first run in the averages to ignore loading effects.
    if (skip_first) {
      skip_first = false;
      continue;
    }
    stats.Update(time);
  }
  return stats;
}

// Calculate the optimimal num_blocks and block_size for the given kernel on
// the given hardware.
void OptimizeOccupancy(hipDeviceProp_t &hardware_info, int &num_blocks,
                       int &block_size, kernelFuncPtr kernel) {
  int min_grid_size;
  hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, kernel, 0, 0);

  int num_blocks_per_SM;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_SM, kernel,
                                                block_size, 0);

  int num_SMs = hardware_info.multiProcessorCount;
  num_blocks = num_blocks_per_SM * num_SMs;

  double current_occupancy =
      Occupancy(hardware_info, num_blocks, block_size, kernel);

  for (int bs = block_size; bs >= 32; bs -= 32) {
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_SM, kernel,
                                                  bs, 0);
    int nb = num_blocks_per_SM * num_SMs;
    double occ = Occupancy(hardware_info, nb, bs, kernel);

    if (occ > current_occupancy) {
      num_blocks = nb;
      block_size = bs;
      current_occupancy = occ;
    }

    if (current_occupancy >= 0.99) break;  // Close enough to 1.0
  }
}

void PrintResults(std::string header, Metrics metrics) {
  Reporter::PrintResults(header + " best      time: ",
                         metrics.get_metrics(Condition::kMinTime));
  Reporter::PrintResults(header + " best  bandwith: ",
                         metrics.get_metrics(Condition::kMaxBandwidth));
  Reporter::PrintResults(header + " best occupancy: ",
                         metrics.get_metrics(Condition::kMaxOccupancy));
}

void RunStrideVariations(hipDeviceProp_t hardware_info, IKernel &ex) {
  Metrics metrics;

  int numBlocks, blockSize;
  OptimizeOccupancy(hardware_info, numBlocks, blockSize, ex.GetKernel());
  std::cout << "expected optimal num_blocks: " << numBlocks << std::endl;
  std::cout << "expected optimal block_size: " << blockSize << std::endl;

  // kFunc<<<num_blocks, block_size>>>
  //                    block_size <= maxThreadsPerBlock
  //         num_blocks <= maxgridsize
  // kFunc<<<max_num_blocks, max_block_size>>>

  // Allocate Unified Memory – accessible from CPU or GPU.
  ex.Setup();
  auto kernel_info = ex.GetKernelInfo();

  auto block_size_gen = ex.GetBlockSizeGenerator();
  while (auto block_size = block_size_gen->Next()) {
    auto num_blocks_gen = ex.GetNumBlocksGenerator();
    while (auto num_blocks = num_blocks_gen->Next()) {
      if (*num_blocks * *block_size > kernel_info.n) {
        *num_blocks = *num_blocks / 2 * 1.1;  // Try just 10% overprovision.
      }
      Reporter::PrintResultsHeader(*num_blocks, *block_size);
      auto occupancy =
          Occupancy(hardware_info, *num_blocks, *block_size, ex.GetKernel());

      auto stats_res =
          RepeatUntil(kRequiredPrecision, ex, *num_blocks, *block_size);

      if (!stats_res) {
        std::cout << " [failed]" << std::endl;
        continue;
      }
      auto mean_res = stats_res->EstimatedMean();
      if (!mean_res || 0.0 == *mean_res) {
        std::cout << " [failed, mean==0.0!]" << std::endl;
        continue;
      }
      auto time_in_ms = *mean_res;
      auto time_in_seconds = time_in_ms / 1000.0;
      auto bandwidth =
          kernel_info.n * kernel_info.bytesPerElement / time_in_seconds;
      Data current_metrics{*num_blocks, *block_size, time_in_ms, bandwidth,
                           occupancy};
      metrics.UpdateAll(current_metrics);
      Reporter::PrintResultsData(current_metrics, stats_res->NumSamples());

      if (*num_blocks * *block_size > kernel_info.n) {
        // n = 1 << 20 = 1,048,576
        // <<<2097152,  1>>> because 2,097,152 *  1 = 2,097,152 > 1,048,576
        // <<<  32768, 64>>> because    32,768 * 64 = 2,097,152 > 1,048,576
        // Try only one overprovision.
        break;
      }
    }
    PrintResults("current", metrics);
  }
  PrintResults("final", metrics);

  ex.Cleanup();
}

int main(void) {
  auto hardware_info = HardwareInfo();
  int max_num_blocks = hardware_info.maxThreadsDim[0] *
                       hardware_info.maxThreadsDim[1] *
                       hardware_info.maxThreadsDim[2];
  int max_block_size = hardware_info.maxThreadsPerBlock;
  std::cout << "max_num_blocks: " << max_num_blocks << std::endl;
  std::cout << "max_block_size: " << max_block_size << std::endl;

  EuclidianDistance ex;
  ex.run();

  Add add(max_num_blocks, max_block_size);
  RunStrideVariations(hardware_info, add);

  return 0;
}
