#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <random>
#include <vector>

#include "add_strided_unmanaged.h"

void AddStridedUnmanaged::Setup() {
  // Allocate memory on the device.
  hipMalloc(&x_, n_ * sizeof(float));
  hipMalloc(&y_, n_ * sizeof(float));

  h_x_ = new float[n_];
  h_y_ = new float[n_];

  for (int j = 0; j < n_; j++) {
    h_x_[j] = 1.0f;
    h_y_[j] = 2.0f;
  }

  // Copy data from host to device.
  hipMemcpy(x_, h_x_, n_ * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_, h_y_, n_ * sizeof(float), hipMemcpyHostToDevice);

  auto err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
  }
}

void AddStridedUnmanaged::RunKernel(int num_blocks, int block_size) {
  AddStridedKernel<<<num_blocks, block_size>>>(n_, x_, y_);
  hipDeviceSynchronize();

  // Copy results back to host.
  hipMemcpy(h_y_, y_, n_ * sizeof(float), hipMemcpyDeviceToHost);
}

void AddStridedUnmanaged::Cleanup() {
  hipFree(x_);
  hipFree(y_);
  delete[] h_x_;
  delete[] h_y_;
}

int AddStridedUnmanaged::CheckResults() {
  int num_errors = 0;
  double max_error = 0.0;

  for (int i = 0; i < n_; i++) {
    if (fabs(h_y_[i] - 3.0f) > 1e-6) {
      num_errors++;
    }
    max_error = fmax(max_error, fabs(h_y_[i] - 3.0f));
  }

  if (num_errors > 0) {
    std::cout << "  number of errors: " << num_errors;
  }
  if (max_error > 0.0) {
    std::cout << ",  max error: " << max_error;
  }

  return num_errors;
}
