#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <random>
#include <vector>

#include "../kernels.h"
#include "add_strided.h"

__global__ void AddStridedKernel(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}

void AddStrided::Setup() {
  hipMallocManaged(&x_, n_ * sizeof(float));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA allocation error: " << hipGetErrorString(err)
              << std::endl;
  }
  hipMallocManaged(&y_, n_ * sizeof(float));
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "allocation error: " << hipGetErrorString(err) << std::endl;
  }
  for (int j = 0; j < n_; j++) {
    x_[j] = 1.0f;
    y_[j] = 2.0f;
  }
}

void AddStrided::RunKernel(int num_blocks, int block_size) {
  AddStridedKernel<<<num_blocks, block_size>>>(n_, x_, y_);
  hipDeviceSynchronize();
}

void AddStrided::Cleanup() {
  hipFree(x_);
  hipFree(y_);
}

int AddStrided::CheckResults() {
  int num_errors = 0;
  double max_error = 0.0;

  for (int i = 0; i < n_; i++) {
    if (fabs(y_[i] - 3.0f) > 1e-6) {
      num_errors++;
    }
    max_error = fmax(max_error, fabs(y_[i] - 3.0f));
  }

  if (num_errors > 0) {
    std::cout << "  number of errors: " << num_errors;
  }
  if (max_error > 0.0) {
    std::cout << ",  max error: " << max_error;
  }

  return num_errors;
}
